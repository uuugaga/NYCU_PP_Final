#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include "kernel.h"

__global__ void DFTKernel(hipDoubleComplex* input, hipDoubleComplex* output, int M, int N) {
    int u = blockIdx.x * blockDim.x + threadIdx.x;
    int v = blockIdx.y * blockDim.y + threadIdx.y;

    if (u < M && v < N) {
        hipDoubleComplex sum = make_hipDoubleComplex(0, 0);
        for (int x = 0; x < M; ++x) {
            for (int y = 0; y < N; ++y) {
                double angle = 2 * M_PI * ((u * x / (double)M) + (v * y / (double)N));
                hipDoubleComplex expVal = make_hipDoubleComplex(cos(angle), -sin(angle));
                sum = hipCadd(sum, hipCmul(input[x * N + y], expVal));
            }
        }
        output[u * N + v] = sum;
    }
}

__global__ void IDFTKernel(hipDoubleComplex* input, hipDoubleComplex* output, int M, int N) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < M && y < N) {
        hipDoubleComplex sum = make_hipDoubleComplex(0, 0);
        for (int u = 0; u < M; ++u) {
            for (int v = 0; v < N; ++v) {
                double angle = 2 * M_PI * ((u * x / (double)M) + (v * y / (double)N));
                hipDoubleComplex expVal = make_hipDoubleComplex(cos(angle), sin(angle));
                sum = hipCadd(sum, hipCmul(input[u * N + v], expVal));
            }
        }
        output[x * N + y] = hipCdiv(sum, make_hipDoubleComplex(M * N, 0));
    }
}

void RunDFT(hipDoubleComplex* input, hipDoubleComplex* output, int M, int N) {
    hipDoubleComplex *d_input, *d_output;
    size_t size = M * N * sizeof(hipDoubleComplex);

    // Allocate device memory
    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);

    // Copy input data to GPU
    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);

    // Kernel launch configuration
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((M + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    DFTKernel<<<numBlocks, threadsPerBlock>>>(d_input, d_output, M, N);

    // Copy result back to host
    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_input);
    hipFree(d_output);
}

// Function to run IDFT on the GPU
void RunIDFT(hipDoubleComplex* input, hipDoubleComplex* output, int M, int N) {
    hipDoubleComplex *d_input, *d_output;
    size_t size = M * N * sizeof(hipDoubleComplex);

    // Allocate device memory
    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);

    // Copy input data to GPU
    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);

    // Kernel launch configuration
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((M + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    IDFTKernel<<<numBlocks, threadsPerBlock>>>(d_input, d_output, M, N);

    // Copy result back to host
    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_input);
    hipFree(d_output);
}